#include "hip/hip_runtime.h"
#include "../hip/hip_runtime.h"
#include "../handle_error.h"
#include <time.h>

#define N 			8
#define THREADS_PER_BLOCK	2



__global__ void add(int *a, int *b, int *c, int max){
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	int id = index;
	while (id < max){
		c[id] = a[id] + b[id];
		// for the problem larger than the number of threads incrase
		// the index is increased by the total number of threads
		id = id + blockDim.x*gridDim.x;
	}
}

void fill_mat(int *mat){
	int c = 0;
	for (int i = 0; i < N; i++){
		for(int j = 0; j<N; j++){
			mat[j + i*N] = c++;
		}
	}
}

void print_mat(int *mat){
	for (int i = 0; i<N; i++){
		for(int j =0; j<N; j++){
			printf("%i\t", mat[j + i*N]);
		}
		printf("\n");
	}
}

int main(){
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int tam = N*N*sizeof(int);

	HANDLE_ERROR(hipMalloc((void**)&d_a,tam));
	HANDLE_ERROR(hipMalloc((void**)&d_b,tam));
	HANDLE_ERROR(hipMalloc((void**)&d_c,tam));

	a = (int*)malloc(tam);
	b = (int*)malloc(tam);
	c = (int*)malloc(tam);

	fill_mat(a);
	fill_mat(b);

	print_mat(a);
	print_mat(b);
	printf("\n");


	hipMemcpy(d_a,a,tam,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,tam,hipMemcpyHostToDevice);

	add <<<N*N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c, N*N);

	hipMemcpy(c,d_c,tam,hipMemcpyDeviceToHost);

	print_mat(c);


	free(a);
	free(b);
	free(c);
	
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}





	

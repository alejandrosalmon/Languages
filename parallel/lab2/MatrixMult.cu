#include "hip/hip_runtime.h"
// Alejandro Salmon Felix Diaz
// A01201954

//incluir path a propio cuda_runtime
#include "../hip/hip_runtime.h"
#include "handle_error.h"
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 	16

__global__ void mat_mult(float* dmat_res, float* dmat_a, float* dmat_b, int a_row, int a_col, int b_col){
	
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	if(col<b_col && row<a_row){
	float result = 0;
		for(int i=0; i<a_col; i++){
			result += dmat_a[row*a_col+i] * dmat_b[i*b_col+col];
		}
		dmat_res[row*b_col+col] = result;
	}
}

//hur dur
void print_mat(float* mat, int row, int col){
    for (int i = 0; i < row; i++){
        for (int j = 0; j < col; j++){
            printf("%.1f\t", mat[i*col+j]);
        }
        printf("\n");
    }
    printf("\n");
}

//llena matrices igual que en los casos de prueba del lab
void fill_matab(float* mata, int rowa, int cola, float* matb, int rowb, int colb){
	int c = 1;
	for(int i = 0; i<rowa; i++){
		for(int j = 0; j<cola; j++){
			mata[i*cola+j] = c++%10;
		}
	}
	c--;
	for(int i = 0; i<cola; i++){
		for(int j = 0; j<colb; j++){
			matb[i*colb+j] = c++%10;
		}
	}
}


int main(int argc, char* argv[]){
	float *hmat_a, *hmat_b, *hmat_res;
	float *dmat_a, *dmat_b, *dmat_res;

	if (argc != 5) {
		printf("usage: %s [MatrixA Rows] [MatrixA Columns] [MatrixB Rows] [MatrixB Columns]\n", argv[0]);
		return -1;
	}


	//realmente no se utiliza
	int a_row = atoi(argv[1]);
	int a_col = atoi(argv[2]);
	/*
		realmente nunca se usa b_row pero memoria de sobra thats why 
		DISCLAIMER:
		(si el uso innecesario de memora molesta al lector favor de comentar la linea siguiente)
	*/
	int b_row = atoi(argv[3]);
	int b_col = atoi(argv[4]);

	if(a_col != atoi(argv[3])){
		printf("Matrix dimensions are not correct\n");
		 	 return -1;
	}


	//genera matrices para producto punto
	hmat_a = (float *)malloc(sizeof(float)*a_row*a_col);
	hmat_b = (float *)malloc(sizeof(float)*a_col*b_col);
	hmat_res = (float *)malloc(sizeof(float)*a_row*a_row);

	fill_matab(hmat_a, a_row, a_col, hmat_b, a_col, b_col);

	print_mat(hmat_a, a_row, a_col);
	print_mat(hmat_b, a_col, b_col);

	HANDLE_ERROR(hipMalloc((void**)&dmat_a,sizeof(float)*a_row*a_col));
	HANDLE_ERROR(hipMalloc((void**)&dmat_b,sizeof(float)*b_row*b_col));
	HANDLE_ERROR(hipMalloc((void**)&dmat_res,sizeof(float)*a_row*a_row));


    hipMemcpy(dmat_a, hmat_a, sizeof(float)*a_row*a_col, hipMemcpyHostToDevice);
    hipMemcpy(dmat_b, hmat_b, sizeof(float)*a_col*b_col, hipMemcpyHostToDevice);


    // Funciones que encontre en linea para calcular bloques y threads optimos dependiendo del tamano de las matrices
    dim3 dimGrid((b_col + BLOCK_SIZE - 1) / BLOCK_SIZE, (a_row + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    //manda llamar la funcion de GPU
    mat_mult<<<dimGrid, dimBlock>>>(dmat_res, dmat_a, dmat_b, a_row, a_col, b_col);

    //copia matriz resultante
    hipMemcpy(hmat_res, dmat_res, sizeof(float)*a_row*a_row, hipMemcpyDeviceToHost);

    //imprime resultado
    print_mat(hmat_res, a_row, a_row);


    //libera memoria de host
    free(hmat_a);
	free(hmat_b);
    free(hmat_res);
    //libera memoria de device
    hipFree(dmat_a);
    hipFree(dmat_b);
    hipFree(dmat_res);

}
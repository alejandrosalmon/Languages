
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 11
#define M 11
#define ThreadsPerBlock 10
#define NumBlocks  (ThreadsPerBlock + (N*M-1))/ThreadsPerBlock

__device__ void convolution(int conv_col, int conv_row, float *d_kernel, int k_size, float *d_matrix, int size_x, int size_y, float *d_conv, int max_row, int max_col){
    int conv_index = conv_col+ conv_row*max_col;
    d_conv[conv_index] = 0;
    for(int k_row = 0;  k_row < k_size; k_row ++){
            for(int k_col = 0;  k_col < k_size ; k_col ++){
                d_conv[conv_index] += 
                d_kernel[k_col + (k_row*k_size)] *
                d_matrix[(conv_col+k_col) + (conv_row+k_row)*size_x];
                //printf("row %i col %i d_conv[] = %f \n", k_row, k_col, d_conv[k_col+ k_row*max_col]);
            }
        }
}

__global__ void valid_convolution(float *d_kernel, int k_size, float *d_matrix, int size_x, int size_y, float *d_conv, int max_row, int max_col){
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
   
    if(max_row > row && max_col > col){
        convolution(col, row, d_kernel, k_size, d_matrix, size_x, size_y, d_conv, max_row, max_col);
    }
}

void print_mat(float *mat, int n){
    for (int i = 0; i < n; i++){
        for (int j = 0; j < n; j++){
            printf("%.1f\t", mat[i*n+j]);
        }
        printf("\n");
    }
    printf("\n");
}


void fill_mat(float *mat, int n){
    int c = 0;
    for (int i = 0; i < n; i++){
        for (int j = 0; j < n; j++){
            mat[i*n+j] = c++%10;
        }
    }
}

void fill_ker(float *mat, int n){
    float c = 1;
    for (int i = 0; i < n; i++){
        for (int j = 0; j < n; j++){
        	//mat[i*n+j] = 0;
        }
    }   
    mat[2*n+3] = 1; 
}

int main(){
    float *h_kernel, *h_matrix, *h_conv;
    float *d_kernel, *d_matrix, *d_conv;

    int k_size = 5;
    int size_x = N;
    int size_y = M;
    int max_row = size_x - (k_size/2)*2;
    int max_col = size_y - (k_size/2)*2;

    h_kernel = (float *)malloc(sizeof(float)*k_size*k_size);
    h_matrix = (float *)malloc(sizeof(float)*size_x*size_y);
    h_conv = (float *)malloc(sizeof(float)*max_row*max_col);

    fill_ker(h_kernel, k_size);
    fill_mat(h_matrix, size_x);

    printf("\n\n----------- KERNEL/FILTRO A APLICAR: \n");
    print_mat(h_kernel, k_size);
    printf("\n\n----------- MATRIZ ORIGINAL: \n");
    print_mat(h_matrix, size_x);

   
    hipMalloc((void**)&d_kernel,sizeof(float)*k_size*k_size);
    hipMalloc((void**)&d_matrix,sizeof(float)*size_x*size_y);
    hipMalloc((void**)&d_conv,sizeof(float)*max_row*max_col);

    hipMemcpy(d_kernel, h_kernel,sizeof(float)*k_size*k_size, hipMemcpyHostToDevice);
    hipMemcpy(d_matrix, h_matrix,sizeof(float)*size_x*size_y, hipMemcpyHostToDevice);

    dim3 Blocks(NumBlocks,NumBlocks);
    dim3 Threads(ThreadsPerBlock,ThreadsPerBlock);
   
    //printf("Blocks %i \nThreads %i \n", NumBlocks, ThreadsPerBlock);
    valid_convolution<<<Blocks, Threads>>>(d_kernel, k_size, d_matrix, size_x, size_y, d_conv, max_row, max_col);
 
    hipMemcpy(h_conv, d_conv,sizeof(float)*max_row*max_col, hipMemcpyDeviceToHost);

    printf("\n\n------------ MATRIZ RESULTANTE: \n");
    print_mat(h_conv, max_col);
   
    free(h_kernel);
    free(h_conv);
    free(h_matrix);
   
    hipFree(d_kernel);
    hipFree(d_conv);
    hipFree(d_matrix);
}

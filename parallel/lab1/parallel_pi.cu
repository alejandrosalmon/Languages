#include "hip/hip_runtime.h"
﻿/*
   Alejandro Salmon F.D.
   A01201954
   Lab1: Parallel PI
*/

// inlude path of your own hip/hip_runtime.h... if its in the same folder just delete the ../
#include "../hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#define NO_RECTANGLES 1000000000
#define width 1.0/NO_RECTANGLES

#define BLOCKS_PER_GRID 12
#define THREADS_PER_BLOCK 512

// Graphic processor calculation of Pi in parallel way
__global__ void parallel_Pi(double *device_array) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int aux = tid;
	double mid;
	double height;
	while (aux < NO_RECTANGLES) {
		mid = (aux + 0.5) * width;
		height = 4.0 / (1.0 + mid * mid);
		device_array[tid] += height;
		aux += blockDim.x * gridDim.x;
	}
}

int main(void) {

	int size = THREADS_PER_BLOCK * BLOCKS_PER_GRID;

    // Device array and local array for getting pi values from gpu and copying them to host
    double *local_array;
	double *device_array;

	//allocate local memory to get values from gpu
	local_array = (double*) malloc(size * sizeof(double));
	// allocate memory in device (GPU)
	hipMalloc((void**) &device_array, size * sizeof(double));


	parallel_Pi<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(device_array);

	// copy result from device to host so we can handle it
	hipMemcpy(local_array, device_array,  size * sizeof(double), hipMemcpyDeviceToHost);

	double pi = 0;

	// Pi is sum of all values in local array multiplied by the specified width of the rectangles
	for(int i = 0; i < size; i++) {
		pi += local_array[i];
	}
	pi = pi * width;

	printf("Calculated pi value: %lf\n", pi);
	printf("YAY PARALLEL\n");

    // Free memory space in device and host
    free(local_array);
	hipFree(device_array);

	return 0;
}

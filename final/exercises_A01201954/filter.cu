
#include <hip/hip_runtime.h>
#include <stdio.h>
#define ThreadsPerBlock 10

__device__ void convolution(int conv_col, int conv_row, float *d_kernel, int k_size, float *d_matrix, int size_x, int size_y, float *d_conv, int max_row, int max_col){
    int conv_index = conv_col+ conv_row*max_col;
    d_conv[conv_index] = 0;
    for(int k_row = 0;  k_row < k_size; k_row ++){
            for(int k_col = 0;  k_col < k_size ; k_col ++){
                d_conv[conv_index] += 
                d_kernel[k_col + (k_row*k_size)] *
                d_matrix[(conv_col+k_col) + (conv_row+k_row)*size_x];
            }
        }
}

__global__ void convolute(float *d_kernel, int k_size, float *d_matrix, int size_x, int size_y, float *d_conv, int max_row, int max_col){
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    //flattens matrix to be able to process it properly
    if(max_row > row && max_col > col){
        convolution(col, row, d_kernel, k_size, d_matrix, size_x, size_y, d_conv, max_row, max_col);
    }
}

void print_mat(float *mat, int n){
    for (int i = 0; i < n; i++){
        for (int j = 0; j < n; j++){
            printf("%.1f\t", mat[i*n+j]);
        }
        printf("\n");
    }
    printf("\n");
}


void fill_mat(float *mat, int n){
    int c = 0;
    for (int i = 0; i < n; i++){
        for (int j = 0; j < n; j++){
            //mat[i*n+j] = c++%10;
	    mat[i*n+j] = 2;
        }
    }
}

void fill_ker(float *mat){
    int size = 3;
    for (int i = 0; i < size; i++){
        for (int j = 0; j < size; j++){
            if(i==1 && j == 1){
                mat[i*size+j] = 8;
            } else {
        	   mat[i*size+j] = -1;
            }
        }
    }    
}

int main(){
    float *h_kernel, *h_matrix, *h_conv;
    float *d_kernel, *d_matrix, *d_conv;

    int k_size = 3;
    int size_x, size_y;

    printf("Please enter the size of the square matrix to convolute over: \n");
    scanf("%d", &size_x);

    size_y = size_x;


    int max_row = size_x - (k_size/2)*2;
    int max_col = size_y - (k_size/2)*2;

    int numBlocks = (ThreadsPerBlock + (size_x*size_y-1))/ThreadsPerBlock;

    h_kernel = (float *)malloc(sizeof(float)*k_size*k_size);
    h_matrix = (float *)malloc(sizeof(float)*size_x*size_y);
    h_conv = (float *)malloc(sizeof(float)*max_row*max_col);

    fill_ker(h_kernel);
    fill_mat(h_matrix, size_x);

    printf("\n\n----------- Kernel to apply: \n");
    print_mat(h_kernel, k_size);
    printf("\n\n----------- Original Matrix to convolute: \n");
    print_mat(h_matrix, size_x);

   
    hipMalloc((void**)&d_kernel,sizeof(float)*k_size*k_size);
    hipMalloc((void**)&d_matrix,sizeof(float)*size_x*size_y);
    hipMalloc((void**)&d_conv,sizeof(float)*max_row*max_col);

    hipMemcpy(d_kernel, h_kernel,sizeof(float)*k_size*k_size, hipMemcpyHostToDevice);
    hipMemcpy(d_matrix, h_matrix,sizeof(float)*size_x*size_y, hipMemcpyHostToDevice);

    dim3 Blocks(numBlocks,numBlocks);
    dim3 Threads(ThreadsPerBlock,ThreadsPerBlock);
   
    //printf("Blocks %i \nThreads %i \n", numBlocks, ThreadsPerBlock);
    convolute<<<Blocks, Threads>>>(d_kernel, k_size, d_matrix, size_x, size_y, d_conv, max_row, max_col);
 
    hipMemcpy(h_conv, d_conv,sizeof(float)*max_row*max_col, hipMemcpyDeviceToHost);

    printf("\n\n------------ Resulting Matrix: \n");
    print_mat(h_conv, max_col);
   
    free(h_kernel);
    free(h_conv);
    free(h_matrix);
   
    hipFree(d_kernel);
    hipFree(d_conv);
    hipFree(d_matrix);
}
